#include "hip/hip_runtime.h"
// Copyright Alan (AJ) Pryor, Jr. 2017
// Transcribed from MATLAB code by Colin Ophus
// Prismatic is distributed under the GNU General Public License (GPL)
// If you use Prismatic, we kindly ask that you cite the following papers:

// 1. Ophus, C.: A fast image simulation algorithm for scanning
//    transmission electron microscopy. Advanced Structural and
//    Chemical Imaging 3(1), 13 (2017)

// 2. Pryor, Jr., A., Ophus, C., and Miao, J.: A Streaming Multi-GPU
//    Implementation of Image Simulation Algorithms for Scanning
//	  Transmission Electron Microscopy. arXiv:1706.08563 (2017)

#include "utility.cuh"
#include "utility.h"
#include "hip/hip_complex.h"
#include <iostream>
#include <sstream>
#include <mutex>

// std::mutex HDF5_lock;

#define PI 3.14159265359
// define some constants
__device__ __constant__ float pi_f                  = PI;
__device__ __constant__ hipFloatComplex i_f          = {0, 1};
__device__ __constant__ hipFloatComplex pi_cx_f      = {PI, 0};
__device__ __constant__ hipFloatComplex minus_2pii_f = {0, -2*PI};
__device__ __constant__ double pi                   = PI;
__device__ __constant__ hipDoubleComplex i           = {0, 1};
__device__ __constant__ hipDoubleComplex pi_cx       = {PI, 0};
__device__ __constant__ hipDoubleComplex minus_2pii  = {0, -2*PI};

//atomicAdd for doubles on devices with compute capability < 6. This is directly copied from the CUDA Programming Guide
#if __CUDA_ARCH__ < 600
__device__  double atomicAdd_double(double* address, const double val)
{
	unsigned long long int* address_as_ull =
			(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;

	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
		                __double_as_longlong(val +
		                                     __longlong_as_double(assumed)));

//		 Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
	} while (assumed != old);

	return __longlong_as_double(old);
}
#endif



// computes exp(real(a) + i * imag(a))
__device__ __forceinline__ hipDoubleComplex exp_cx(const hipDoubleComplex a){
	double e = exp(a.x);
	double s,c;
	sincos(a.y, &s, &c);
	return make_hipDoubleComplex(e*c, e*s);
}
__device__ __forceinline__ hipFloatComplex exp_cx(const hipFloatComplex a){
	float e = expf(a.x);
	float s,c;
	sincosf(a.y, &s, &c);
	return make_hipFloatComplex(e*c, e*s);
}

__global__ void initializePsi_oneNonzero(hipFloatComplex *psi_d, const size_t N, const size_t beamLoc){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		psi_d[idx] = (idx == beamLoc) ? make_hipFloatComplex(1,0):make_hipFloatComplex(0,0);
	}
}

__global__ void initializePsi_oneNonzero(hipDoubleComplex *psi_d, const size_t N, const size_t beamLoc){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		psi_d[idx] = (idx == beamLoc) ? make_hipDoubleComplex(1,0):make_hipDoubleComplex(0,0);
	}
}

// multiply two complex arrays
__global__ void multiply_inplace(hipDoubleComplex* arr,
                                 const hipDoubleComplex* other,
                                 const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		hipDoubleComplex a = arr[idx];
		hipDoubleComplex o = other[idx];
		arr[idx].x = a.x * o.x - a.y * o.y;
		arr[idx].y = a.x * o.y + a.y * o.x;
	}
}

// multiply two complex arrays
__global__ void multiply_inplace(hipFloatComplex* arr,
                                 const hipFloatComplex* other,
                                 const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		hipFloatComplex a = arr[idx];
		hipFloatComplex o = other[idx];
		arr[idx].x = a.x * o.x - a.y * o.y;
		arr[idx].y = a.x * o.y + a.y * o.x;
	}
}

// multiply two complex arrays
__global__ void multiply_cx(hipDoubleComplex* arr,
                             const hipDoubleComplex* other,
                             const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
//		hipDoubleComplex a = arr[idx];
//		hipDoubleComplex o = other[idx];
		arr[idx] = hipCmul(arr[idx], other[idx]);
	}
}

// multiply two complex arrays
__global__ void multiply_cx(hipFloatComplex* arr,
                            const hipFloatComplex* other,
                            const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = hipCmulf(arr[idx], other[idx]);
	}
}

// multiply complex array by scalar
__global__ void multiply_cxarr_scalar(hipDoubleComplex* arr,
                                      const hipDoubleComplex val,
                                      const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = hipCmul(arr[idx], val);
	}
}

// multiply complex array by scalar
__global__ void multiply_cxarr_scalar(hipFloatComplex* arr,
                                      const hipFloatComplex val,
                                      const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = hipCmulf(arr[idx], val);
	}
}

// multiply array by scalar
__global__ void multiply_arr_scalar(double* arr,
                                    const double val,
                                    const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = arr[idx]*val;
	}
}

// multiply array by scalar
__global__ void multiply_arr_scalar(float* arr,
                                    const float val,
                                    const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = arr[idx]*val;
	}
}


//// divide two complex arrays
//__global__ void divide_inplace(PRISMATIC_CUDA_COMPLEX_FLOAT* arr,
//                               const PRISMATIC_FLOAT_PRECISION val,
//                               const size_t N){
//	int idx = threadIdx.x + blockDim.x*blockIdx.x;
//	if (idx < N) {
//		arr[idx].x /= val;
//		arr[idx].y /= val;
//	}
//}

__global__ void divide_inplace(hipDoubleComplex* arr,
                               const hipDoubleComplex val,
                               const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = hipCdiv(arr[idx], val);
	}
}

__global__ void divide_inplace(hipFloatComplex* arr,
                               const hipFloatComplex val,
                               const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = hipCdivf(arr[idx], val);
	}
}

// set all array values to val
__global__ void setAll(double *data, double val, size_t N) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx<N) {
		data[idx] = val;
	}
}

// set all array values to val
__global__ void setAll(float *data, float val, size_t N) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx<N) {
		data[idx] = val;
	}
}

// creates initial probe using existing GPU memory rather than streaming each probe
__global__ void initializePsi(hipDoubleComplex *psi_d,
                              const hipDoubleComplex* PsiProbeInit_d,
                              const double* qya_d,
                              const double* qxa_d,
                              const size_t N,
                              const double yp,
                              const double xp){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		hipDoubleComplex arg;
		arg = make_hipDoubleComplex(qxa_d[idx]*xp + qya_d[idx]*yp, 0);
		psi_d[idx] = hipCmul(PsiProbeInit_d[idx], exp_cx(hipCmul(minus_2pii,arg)));
	}
}

// creates initial probe using existing GPU memory rather than streaming each probe
__global__ void initializePsi(hipFloatComplex *psi_d,
                              const hipFloatComplex* PsiProbeInit_d,
                              const float* qya_d,
                              const float* qxa_d,
                              const size_t N,
                              const float yp,
                              const float xp){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		hipFloatComplex arg;
		arg = make_hipFloatComplex(qxa_d[idx]*xp + qya_d[idx]*yp, 0);
		psi_d[idx] = hipCmulf(PsiProbeInit_d[idx], exp_cx(hipCmulf(minus_2pii_f,arg)));
	}
}


// compute modulus squared of other and store in arr
__global__ void abs_squared(double* arr,
                            const hipDoubleComplex* other,
                            const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		double re = other[idx].x;
		double im = other[idx].y;
		arr[idx] = re*re + im*im;
	}
}

// compute modulus squared of other and store in arr
__global__ void abs_squared(float* arr,
                            const hipFloatComplex* other,
                            const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		float re = other[idx].x;
		float im = other[idx].y;
		arr[idx] = re*re + im*im;
	}
}

__global__ void array_subset(const hipDoubleComplex* psi_d,
                             hipDoubleComplex* psi_small_d,
                             const size_t* qyInd_d,
                             const size_t* qxInd_d,
                             const size_t dimi,
                             const size_t dimj_small,
                             const size_t dimi_small){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < dimj_small*dimi_small) {
		int y = idx / (int)dimi_small;
		int x = idx % (int)dimi_small;
		int idxBig = qyInd_d[y] * dimi + qxInd_d[x];
		psi_small_d[idx] = psi_d[idxBig];
//		psi_small_d[idx] = make_hipFloatComplex(idx,idxBig);
	}
}
__global__ void array_subset(const hipFloatComplex* psi_d,
                             hipFloatComplex* psi_small_d,
                             const size_t* qyInd_d,
                             const size_t* qxInd_d,
                             const size_t dimi,
                             const size_t dimj_small,
                             const size_t dimi_small) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < dimj_small * dimi_small) {
		int y = idx / (int) dimi_small;
		int x = idx % (int) dimi_small;
		int idxBig = qyInd_d[y] * dimi + qxInd_d[x];
		psi_small_d[idx] = psi_d[idxBig];
//		psi_small_d[idx] = make_hipFloatComplex(idx,idxBig);
	}
}




__global__ void shiftIndices(long* vec_out, const long by, const long imageSize, const long N){

		//int idx = threadIdx.x + blockDim.x * blockIdx.x;
		long idx = threadIdx.x + blockDim.x * blockIdx.x;
		if (idx < N){
			vec_out[idx] = (imageSize + ((idx - N/2 + by) % imageSize)) % imageSize;
//			vec_out[idx] =  (idx - N/2 + by) % imageSize;
//			vec_out[idx] = 0;
		}
	}

__global__ void zeroIndices(long* vec_out, const long N){

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N){
		vec_out[idx] = vec_out[idx] - vec_out[0];
	}
}

__global__ void resetIndices(long* vec_out, const long N){

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N){
		vec_out[idx] = idx;
	}
}


__global__ void computePhaseCoeffs(hipFloatComplex* phaseCoeffs,
                                   const hipFloatComplex *PsiProbeInit_d,
                                   const float * qyaReduce_d,
                                   const float * qxaReduce_d,
                                   const size_t *yBeams_d,
                                   const size_t *xBeams_d,
                                   const float yp,
                                   const float xp,
                                   const float yTiltShift,
                                   const float xTiltShift,
                                   const size_t dimi,
                                   const size_t numBeams){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < numBeams) {
		size_t yB = yBeams_d[idx];
		size_t xB = xBeams_d[idx];
		hipFloatComplex xp_cx = make_hipFloatComplex(xp, 0);
		hipFloatComplex yp_cx = make_hipFloatComplex(yp, 0);
		hipFloatComplex xTiltShift_cx = make_hipFloatComplex(xTiltShift, 0);
		hipFloatComplex yTiltShift_cx = make_hipFloatComplex(yTiltShift, 0);
		hipFloatComplex qya = make_hipFloatComplex(qyaReduce_d[yB * dimi + xB], 0);
		hipFloatComplex qxa = make_hipFloatComplex(qxaReduce_d[yB * dimi + xB], 0);
		hipFloatComplex arg1 = hipCmulf(qxa, hipCaddf(xp_cx, xTiltShift_cx));
		hipFloatComplex arg2 = hipCmulf(qya, hipCaddf(yp_cx, yTiltShift_cx));
		hipFloatComplex arg = hipCaddf(arg1, arg2);
		hipFloatComplex phase_shift = exp_cx(hipCmulf(minus_2pii_f, arg));
		phaseCoeffs[idx] = hipCmulf(phase_shift, PsiProbeInit_d[yB * dimi + xB]);
	}
}

__global__ void computePhaseCoeffs(hipDoubleComplex* phaseCoeffs,
                                   const hipDoubleComplex *PsiProbeInit_d,
                                   const double * qyaReduce_d,
                                   const double * qxaReduce_d,
                                   const size_t *yBeams_d,
                                   const size_t *xBeams_d,
                                   const double yp,
                                   const double xp,
                                   const double yTiltShift,
                                   const double xTiltShift,
                                   const size_t dimi,
                                   const size_t numBeams){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < numBeams) {
		size_t yB = yBeams_d[idx];
		size_t xB = xBeams_d[idx];
		hipDoubleComplex xp_cx = make_hipDoubleComplex(xp, 0);
		hipDoubleComplex yp_cx = make_hipDoubleComplex(yp, 0);
		hipDoubleComplex xTiltShift_cx = make_hipDoubleComplex(xTiltShift, 0);
		hipDoubleComplex yTiltShift_cx = make_hipDoubleComplex(yTiltShift, 0);
		hipDoubleComplex qya = make_hipDoubleComplex(qyaReduce_d[yB * dimi + xB], 0);
		hipDoubleComplex qxa = make_hipDoubleComplex(qxaReduce_d[yB * dimi + xB], 0);
		hipDoubleComplex arg1 = hipCmul(qxa, hipCadd(xp_cx, xTiltShift_cx));
		hipDoubleComplex arg2 = hipCmul(qya, hipCadd(yp_cx, yTiltShift_cx));
		hipDoubleComplex arg = hipCadd(arg1, arg2);
		hipDoubleComplex phase_shift = exp_cx(hipCmul(minus_2pii, arg));
		phaseCoeffs[idx] = hipCmul(phase_shift, PsiProbeInit_d[yB * dimi + xB]);
	}
}





// integrate computed intensities radially
__global__ void integrateDetector(const float* psiIntensity_ds,
                                  const float* alphaInd_d,
                                  float* integratedOutput,
                                  const size_t N,
                                  const size_t num_integration_bins) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N) {
		size_t alpha = (size_t)alphaInd_d[idx];
		if (alpha <= num_integration_bins)
			//atomicAdd(&integratedOutput[alpha-1], psiIntensity_ds[idx]);
			atomicAdd(&integratedOutput[alpha-1], psiIntensity_ds[idx]);
	}
}

__global__ void integrateDetector(const double* psiIntensity_ds,
                                  const double* alphaInd_d,
                                  double* integratedOutput,
                                  const size_t N,
                                  const size_t num_integration_bins) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N) {
		size_t alpha = (size_t)alphaInd_d[idx];
		if (alpha <= num_integration_bins)
			//atomicAdd(&integratedOutput[alpha-1], psiIntensity_ds[idx]);
			atomicAdd_double(&integratedOutput[alpha-1], psiIntensity_ds[idx]);
	}
}

__global__ void DPC_numerator_reduce(const float* psiIntensity_ds,
									 const float* q_coord,
									 float* numerator,
									 const size_t N){
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N){
		atomicAdd(&numerator[0], psiIntensity_ds[idx]*q_coord[idx]);
	}
}

__global__ void DPC_numerator_reduce(const double* psiIntensity_ds,
 									 const double* q_coord,
									 double* numerator,
									 const size_t N){
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
		if (idx < N){
			atomicAdd_double(&numerator[0], psiIntensity_ds[idx]*q_coord[idx]);
		}
}

__global__ void DPC_denominator_reduce(const float* psiIntensity_ds,
									   float* denominator,
									   const size_t N){
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N){
		atomicAdd(&denominator[0], psiIntensity_ds[idx]);
	}										   
}

__global__ void DPC_denominator_reduce(const double* psiIntensity_ds,
									   double* denominator,
									   const size_t N){
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N){
		atomicAdd_double(&denominator[0], psiIntensity_ds[idx]);
	}			
}

void formatOutput_GPU_integrate(Prismatic::Parameters<PRISMATIC_FLOAT_PRECISION> &pars,
                                PRISMATIC_FLOAT_PRECISION *psiIntensity_ds,
                                const PRISMATIC_FLOAT_PRECISION *alphaInd_d,
                                PRISMATIC_FLOAT_PRECISION *output_ph,
								PRISMATIC_FLOAT_PRECISION *integratedOutput_ds,
								const PRISMATIC_FLOAT_PRECISION* qya_d,
								const PRISMATIC_FLOAT_PRECISION* qxa_d,
								const size_t currentSlice,
                                const size_t ay,
                                const size_t ax,
                                const size_t& dimj,
                                const size_t& dimi,
                                const hipStream_t& stream,
                                const long& scale) {

	//save 4D output if applicable
    if (pars.meta.save4DOutput)
    {
		// This section could be improved. It currently makes a new 2D array, copies to it, and
		// then saves the image. This allocates arrays multiple times unneccessarily, and the allocated
		// memory isn't pinned, so the memcpy is not asynchronous.
		//std::string section4DFilename = generateFilename(pars, currentSlice, ay, ax);
		
		Prismatic::Array2D<PRISMATIC_FLOAT_PRECISION> currentImage = Prismatic::zeros_ND<2, PRISMATIC_FLOAT_PRECISION>(
				{{pars.psiProbeInit.get_dimj(), pars.psiProbeInit.get_dimi()}});
		cudaErrchk(hipMemcpyAsync(&currentImage[0],
		                           psiIntensity_ds,
		                           pars.psiProbeInit.size() * sizeof(PRISMATIC_FLOAT_PRECISION),
		                           hipMemcpyDeviceToHost,
								   stream));
								   
		//Need to scale the output by the square of the PRISM interpolation factor 
		// std::unique_lock<std::mutex> HDF5_gatekeeper(Prismatic::HDF5_lock);

		currentImage *= pars.scale;
		std::stringstream nameString;
		nameString << "/4DSTEM_simulation/data/datacubes/CBED_array_depth" << Prismatic::getDigitString(currentSlice);
		
		// H5::Group dataGroup = pars.outputFile.openGroup(nameString.str());
		// H5::DataSet CBED_data = dataGroup.openDataSet("datacube");

		hsize_t offset[4] = {ax,ay,0,0}; //order by ax, ay so that aligns with py4DSTEM
        PRISMATIC_FLOAT_PRECISION numFP = pars.meta.numFP;
        
        if(pars.meta.crop4DOutput)
        {
            Prismatic::Array2D<PRISMATIC_FLOAT_PRECISION> finalImage = cropOutput(currentImage,pars);
            hsize_t mdims[4] = {1,1,finalImage.get_dimi(),finalImage.get_dimj()};
            Prismatic::writeDatacube4D(pars, &finalImage[0],mdims,offset,numFP,nameString.str());
        }
        else
        {

            if (pars.meta.algorithm == Prismatic::Algorithm::Multislice){
                Prismatic::Array2D<PRISMATIC_FLOAT_PRECISION> finalImage = Prismatic::zeros_ND<2, PRISMATIC_FLOAT_PRECISION>(
                    {{pars.psiProbeInit.get_dimj()/2,pars.psiProbeInit.get_dimi()/2}});
                    {
                        long offset_x = pars.psiProbeInit.get_dimi() / 4;
                        long offset_y = pars.psiProbeInit.get_dimj() / 4;
                        long ndimy = (long) pars.psiProbeInit.get_dimj();
                        long ndimx = (long) pars.psiProbeInit.get_dimi();
                        for (long y = 0; y < pars.psiProbeInit.get_dimj() / 2; ++y) {
                            for (long x = 0; x < pars.psiProbeInit.get_dimi() / 2; ++x) {
                                finalImage.at(y, x) = currentImage.at(((y - offset_y) % ndimy + ndimy) % ndimy,
                                ((x - offset_x) % ndimx + ndimx) % ndimx);
                            }
                        }
                    }
                    
                    //finalImage = fftshift2(finalImage);
                    hsize_t mdims[4] = {1,1,pars.psiProbeInit.get_dimi()/2,pars.psiProbeInit.get_dimj()/2};
                    Prismatic::writeDatacube4D(pars, &finalImage[0],mdims,offset,numFP,nameString.str());
                    //finalImage.toMRC_f(section4DFilename.c_str());
                }else{                     
                    currentImage = fftshift2(currentImage);
                    hsize_t mdims[4] = {1,1,pars.psiProbeInit.get_dimi(),pars.psiProbeInit.get_dimj()};
                    Prismatic::writeDatacube4D(pars, &currentImage[0],mdims,offset,numFP,nameString.str());
                    //currentImage.toMRC_f(section4DFilename.c_str());
                }
        }
        // CBED_data.close();
        // dataGroup.close();
        // HDF5_gatekeeper.unlock();
    }
//		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);


	size_t num_integration_bins = pars.detectorAngles.size();
	setAll <<< (num_integration_bins - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >>>
	                                                                            (integratedOutput_ds, 0, num_integration_bins);

	integrateDetector <<< (dimj * dimi - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >>>
	                                                                              (psiIntensity_ds, alphaInd_d, integratedOutput_ds,
			                                                                              dimj *
			                                                                              dimi, num_integration_bins);

	multiply_arr_scalar <<< (dimj * dimi - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >>>
	                                                                                (integratedOutput_ds, scale, num_integration_bins);

	cudaErrchk(hipMemcpyAsync(output_ph, integratedOutput_ds,
	                           num_integration_bins * sizeof(PRISMATIC_FLOAT_PRECISION),
	                           hipMemcpyDeviceToHost, stream));

	//	 wait for the copy to complete and then copy on the host. Other host threads exist doing work so this wait isn't costing anything
	cudaErrchk(hipStreamSynchronize(stream));
	const size_t stack_start_offset =
			currentSlice * pars.output.get_dimk() * pars.output.get_dimj() * pars.output.get_dimi() + ay * pars.output.get_dimj() * pars.output.get_dimi() + ax * pars.output.get_dimi();
	memcpy(&pars.output[stack_start_offset], output_ph, num_integration_bins * sizeof(PRISMATIC_FLOAT_PRECISION));
	
    if(pars.meta.saveDPC_CoM)
    {
		//device variables
		PRISMATIC_FLOAT_PRECISION *num_qx_d;
		PRISMATIC_FLOAT_PRECISION *num_qy_d;
		PRISMATIC_FLOAT_PRECISION *denominator_d;
		cudaErrchk(hipMallocManaged(&num_qx_d, 1*sizeof(PRISMATIC_FLOAT_PRECISION)));
		cudaErrchk(hipMallocManaged(&num_qy_d, 1*sizeof(PRISMATIC_FLOAT_PRECISION)));
		cudaErrchk(hipMallocManaged(&denominator_d, 1*sizeof(PRISMATIC_FLOAT_PRECISION)));

		//host variables
		PRISMATIC_FLOAT_PRECISION *num_qx_h = new PRISMATIC_FLOAT_PRECISION[1];
		PRISMATIC_FLOAT_PRECISION *num_qy_h = new PRISMATIC_FLOAT_PRECISION[1];
		PRISMATIC_FLOAT_PRECISION *denominator_h = new PRISMATIC_FLOAT_PRECISION[1];
		num_qx_h[0] = 0.0;
		num_qy_h[0] = 0.0;
		denominator_h[0] = 0.0;

		//initialize device variables
		cudaErrchk(hipMemcpyAsync(num_qx_d,&num_qx_h[0],1*sizeof(PRISMATIC_FLOAT_PRECISION),hipMemcpyHostToDevice));
		cudaErrchk(hipMemcpyAsync(num_qy_d,&num_qy_h[0],1*sizeof(PRISMATIC_FLOAT_PRECISION),hipMemcpyHostToDevice));
		cudaErrchk(hipMemcpyAsync(denominator_d,&denominator_h[0],1*sizeof(PRISMATIC_FLOAT_PRECISION),hipMemcpyHostToDevice));
		
		//reduce in X
		DPC_numerator_reduce <<< (dimj * dimi - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >>>
		(psiIntensity_ds,qxa_d, num_qx_d, dimj * dimi);
		
		//reduce in Y
		DPC_numerator_reduce <<< (dimj * dimi - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >>>
		(psiIntensity_ds,qya_d, num_qy_d, dimj * dimi);
		
		DPC_denominator_reduce <<< (dimj * dimi - 1) / BLOCK_SIZE1D + 1, BLOCK_SIZE1D, 0, stream >>> (psiIntensity_ds, denominator_d, dimj*dimi);
		
		//copy back to host
		cudaErrchk(hipMemcpyAsync(&num_qx_h[0],num_qx_d,1*sizeof(PRISMATIC_FLOAT_PRECISION),hipMemcpyDeviceToHost));
		cudaErrchk(hipMemcpyAsync(&num_qy_h[0],num_qy_d,1*sizeof(PRISMATIC_FLOAT_PRECISION),hipMemcpyDeviceToHost));
		cudaErrchk(hipMemcpyAsync(&denominator_h[0],denominator_d,1*sizeof(PRISMATIC_FLOAT_PRECISION),hipMemcpyDeviceToHost));

		PRISMATIC_FLOAT_PRECISION DPC_CoM[2];
		DPC_CoM[0] = num_qx_h[0]/denominator_h[0]; //measurement at ax,ay of CoM w.r.t. qx
		DPC_CoM[1] = num_qy_h[0]/denominator_h[0]; //measurement at ax,ay of CoM w.r.t. qy

		//copy to memory and free variables
		const size_t dpc_stack_offset = 
				currentSlice*pars.DPC_CoM.get_dimk() * pars.DPC_CoM.get_dimj() * pars.DPC_CoM.get_dimi() + ay * pars.DPC_CoM.get_dimj() * pars.DPC_CoM.get_dimi() + ax * pars.DPC_CoM.get_dimi();
		memcpy(&pars.DPC_CoM[dpc_stack_offset],&DPC_CoM[0],2*sizeof(PRISMATIC_FLOAT_PRECISION));
		cudaErrchk(hipFree(num_qx_d));
		cudaErrchk(hipFree(num_qy_d));
		cudaErrchk(hipFree(denominator_d));
		free(num_qx_h);
		free(num_qy_h);
		free(denominator_h);
		//cudaErrchk(hipMemcpyAsync());
	}
}

size_t getNextPower2(const size_t& val){
	size_t p = 0;
	while (pow(2,p) <= val)++p;
	return p;
}