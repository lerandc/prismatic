#include "hip/hip_runtime.h"
// Copyright Alan (AJ) Pryor, Jr. 2017
// Transcribed from MATLAB code by Colin Ophus
// Prismatic is distributed under the GNU General Public License (GPL)
// If you use Prismatic, we kindly ask that you cite the following papers:

// 1. Ophus, C.: A fast image simulation algorithm for scanning
//    transmission electron microscopy. Advanced Structural and
//    Chemical Imaging 3(1), 13 (2017)

// 2. Pryor, Jr., A., Ophus, C., and Miao, J.: A Streaming Multi-GPU
//    Implementation of Image Simulation Algorithms for Scanning
//	  Transmission Electron Microscopy. arXiv:1706.08563 (2017)

#include "utility.cuh"
#include "utility.h"
#include "hip/hip_complex.h"
#include <iostream>
#include <sstream>
#include <mutex>
#include "fileIO.h"

// std::mutex HDF5_lock;

#define PI 3.14159265359
// define some constants
__device__ __constant__ float pi_f                  = PI;
__device__ __constant__ hipFloatComplex i_f          = {0, 1};
__device__ __constant__ hipFloatComplex pi_cx_f      = {PI, 0};
__device__ __constant__ hipFloatComplex minus_2pii_f = {0, -2*PI};
__device__ __constant__ double pi                   = PI;
__device__ __constant__ hipDoubleComplex i           = {0, 1};
__device__ __constant__ hipDoubleComplex pi_cx       = {PI, 0};
__device__ __constant__ hipDoubleComplex minus_2pii  = {0, -2*PI};

//atomicAdd for doubles on devices with compute capability < 6. This is directly copied from the CUDA Programming Guide
#if __CUDA_ARCH__ < 600
__device__  double atomicAdd_double(double* address, const double val)
{
	unsigned long long int* address_as_ull =
			(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;

	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
		                __double_as_longlong(val +
		                                     __longlong_as_double(assumed)));

//		 Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
	} while (assumed != old);

	return __longlong_as_double(old);
}
#endif



// computes exp(real(a) + i * imag(a))
__device__ __forceinline__ hipDoubleComplex exp_cx(const hipDoubleComplex a){
	double e = exp(a.x);
	double s,c;
	sincos(a.y, &s, &c);
	return make_hipDoubleComplex(e*c, e*s);
}
__device__ __forceinline__ hipFloatComplex exp_cx(const hipFloatComplex a){
	float e = expf(a.x);
	float s,c;
	sincosf(a.y, &s, &c);
	return make_hipFloatComplex(e*c, e*s);
}

__global__ void initializePsi_oneNonzero(hipFloatComplex *psi_d, const size_t N, const size_t beamLoc){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		psi_d[idx] = (idx == beamLoc) ? make_hipFloatComplex(1,0):make_hipFloatComplex(0,0);
	}
}

__global__ void initializePsi_oneNonzero(hipDoubleComplex *psi_d, const size_t N, const size_t beamLoc){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		psi_d[idx] = (idx == beamLoc) ? make_hipDoubleComplex(1,0):make_hipDoubleComplex(0,0);
	}
}

// multiply two complex arrays
__global__ void multiply_inplace(hipDoubleComplex* arr,
                                 const hipDoubleComplex* other,
                                 const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		hipDoubleComplex a = arr[idx];
		hipDoubleComplex o = other[idx];
		arr[idx].x = a.x * o.x - a.y * o.y;
		arr[idx].y = a.x * o.y + a.y * o.x;
	}
}

// multiply two complex arrays
__global__ void multiply_inplace(hipFloatComplex* arr,
                                 const hipFloatComplex* other,
                                 const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		hipFloatComplex a = arr[idx];
		hipFloatComplex o = other[idx];
		arr[idx].x = a.x * o.x - a.y * o.y;
		arr[idx].y = a.x * o.y + a.y * o.x;
	}
}

// multiply two complex arrays
__global__ void multiply_cx(hipDoubleComplex* arr,
                             const hipDoubleComplex* other,
                             const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
//		hipDoubleComplex a = arr[idx];
//		hipDoubleComplex o = other[idx];
		arr[idx] = hipCmul(arr[idx], other[idx]);
	}
}

// multiply two complex arrays
__global__ void multiply_cx(hipFloatComplex* arr,
                            const hipFloatComplex* other,
                            const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = hipCmulf(arr[idx], other[idx]);
	}
}

// multiply complex array by scalar
__global__ void multiply_cxarr_scalar(hipDoubleComplex* arr,
                                      const hipDoubleComplex val,
                                      const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = hipCmul(arr[idx], val);
	}
}

// multiply complex array by scalar
__global__ void multiply_cxarr_scalar(hipFloatComplex* arr,
                                      const hipFloatComplex val,
                                      const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = hipCmulf(arr[idx], val);
	}
}

// multiply array by scalar
__global__ void multiply_arr_scalar(double* arr,
                                    const double val,
                                    const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = arr[idx]*val;
	}
}

// multiply array by scalar
__global__ void multiply_arr_scalar(float* arr,
                                    const float val,
                                    const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = arr[idx]*val;
	}
}


//// divide two complex arrays
//__global__ void divide_inplace(PRISMATIC_CUDA_COMPLEX_FLOAT* arr,
//                               const PRISMATIC_FLOAT_PRECISION val,
//                               const size_t N){
//	int idx = threadIdx.x + blockDim.x*blockIdx.x;
//	if (idx < N) {
//		arr[idx].x /= val;
//		arr[idx].y /= val;
//	}
//}

__global__ void divide_inplace(hipDoubleComplex* arr,
                               const hipDoubleComplex val,
                               const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = hipCdiv(arr[idx], val);
	}
}

__global__ void divide_inplace(hipFloatComplex* arr,
                               const hipFloatComplex val,
                               const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		arr[idx] = hipCdivf(arr[idx], val);
	}
}

// set all array values to val
__global__ void setAll(double *data, double val, size_t N) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx<N) {
		data[idx] = val;
	}
}

// set all array values to val
__global__ void setAll(float *data, float val, size_t N) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx<N) {
		data[idx] = val;
	}
}

// set all array values to val
__global__ void setAll(hipDoubleComplex *data, hipDoubleComplex val, size_t N) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx<N) {
		data[idx] = val;
	}
}

// set all array values to val
__global__ void setAll(hipFloatComplex *data, hipFloatComplex val, size_t N) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx<N) {
		data[idx] = val;
	}
}

// creates initial probe using existing GPU memory rather than streaming each probe
__global__ void initializePsi(hipDoubleComplex *psi_d,
                              const hipDoubleComplex* PsiProbeInit_d,
                              const double* qya_d,
                              const double* qxa_d,
                              const size_t N,
                              const double yp,
                              const double xp){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		hipDoubleComplex arg;
		arg = make_hipDoubleComplex(qxa_d[idx]*xp + qya_d[idx]*yp, 0);
		psi_d[idx] = hipCmul(PsiProbeInit_d[idx], exp_cx(hipCmul(minus_2pii,arg)));
	}
}

// creates initial probe using existing GPU memory rather than streaming each probe
__global__ void initializePsi(hipFloatComplex *psi_d,
                              const hipFloatComplex* PsiProbeInit_d,
                              const float* qya_d,
                              const float* qxa_d,
                              const size_t N,
                              const float yp,
                              const float xp){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		hipFloatComplex arg;
		arg = make_hipFloatComplex(qxa_d[idx]*xp + qya_d[idx]*yp, 0);
		psi_d[idx] = hipCmulf(PsiProbeInit_d[idx], exp_cx(hipCmulf(minus_2pii_f,arg)));
	}
}


// compute modulus squared of other and store in arr
__global__ void abs_squared(double* arr,
                            const hipDoubleComplex* other,
                            const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		double re = other[idx].x;
		double im = other[idx].y;
		arr[idx] = re*re + im*im;
	}
}

// compute modulus squared of other and store in arr
__global__ void abs_squared(float* arr,
                            const hipFloatComplex* other,
                            const size_t N){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < N) {
		float re = other[idx].x;
		float im = other[idx].y;
		arr[idx] = re*re + im*im;
	}
}

__global__ void array_subset(const hipDoubleComplex* psi_d,
                             hipDoubleComplex* psi_small_d,
                             const size_t* qyInd_d,
                             const size_t* qxInd_d,
                             const size_t dimi,
                             const size_t dimj_small,
                             const size_t dimi_small){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < dimj_small*dimi_small) {
		int y = idx / (int)dimi_small;
		int x = idx % (int)dimi_small;
		int idxBig = qyInd_d[y] * dimi + qxInd_d[x];
		psi_small_d[idx] = psi_d[idxBig];
//		psi_small_d[idx] = make_hipFloatComplex(idx,idxBig);
	}
}
__global__ void array_subset(const hipFloatComplex* psi_d,
                             hipFloatComplex* psi_small_d,
                             const size_t* qyInd_d,
                             const size_t* qxInd_d,
                             const size_t dimi,
                             const size_t dimj_small,
                             const size_t dimi_small) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < dimj_small * dimi_small) {
		int y = idx / (int) dimi_small;
		int x = idx % (int) dimi_small;
		int idxBig = qyInd_d[y] * dimi + qxInd_d[x];
		psi_small_d[idx] = psi_d[idxBig];
//		psi_small_d[idx] = make_hipFloatComplex(idx,idxBig);
	}
}




__global__ void shiftIndices(long* vec_out, const long by, const long imageSize, const long N){

		//int idx = threadIdx.x + blockDim.x * blockIdx.x;
		long idx = threadIdx.x + blockDim.x * blockIdx.x;
		if (idx < N){
			vec_out[idx] = (imageSize + ((idx - N/2 + by) % imageSize)) % imageSize;
//			vec_out[idx] =  (idx - N/2 + by) % imageSize;
//			vec_out[idx] = 0;
		}
	}

__global__ void zeroIndices(long* vec_out, const long N){

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N){
		vec_out[idx] = vec_out[idx] - vec_out[0];
	}
}

__global__ void resetIndices(long* vec_out, const long N){

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N){
		vec_out[idx] = idx;
	}
}


__global__ void computePhaseCoeffs(hipFloatComplex* phaseCoeffs,
                                   const hipFloatComplex *PsiProbeInit_d,
                                   const float * qyaReduce_d,
                                   const float * qxaReduce_d,
                                   const size_t *yBeams_d,
                                   const size_t *xBeams_d,
                                   const float yp,
                                   const float xp,
                                   const float yTiltShift,
                                   const float xTiltShift,
                                   const size_t dimi,
                                   const size_t numBeams){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < numBeams) {
		size_t yB = yBeams_d[idx];
		size_t xB = xBeams_d[idx];
		hipFloatComplex xp_cx = make_hipFloatComplex(xp, 0);
		hipFloatComplex yp_cx = make_hipFloatComplex(yp, 0);
		hipFloatComplex xTiltShift_cx = make_hipFloatComplex(xTiltShift, 0);
		hipFloatComplex yTiltShift_cx = make_hipFloatComplex(yTiltShift, 0);
		hipFloatComplex qya = make_hipFloatComplex(qyaReduce_d[yB * dimi + xB], 0);
		hipFloatComplex qxa = make_hipFloatComplex(qxaReduce_d[yB * dimi + xB], 0);
		hipFloatComplex arg1 = hipCmulf(qxa, hipCaddf(xp_cx, xTiltShift_cx));
		hipFloatComplex arg2 = hipCmulf(qya, hipCaddf(yp_cx, yTiltShift_cx));
		hipFloatComplex arg = hipCaddf(arg1, arg2);
		hipFloatComplex phase_shift = exp_cx(hipCmulf(minus_2pii_f, arg));
		phaseCoeffs[idx] = hipCmulf(phase_shift, PsiProbeInit_d[yB * dimi + xB]);
	}
}

__global__ void computePhaseCoeffs(hipDoubleComplex* phaseCoeffs,
                                   const hipDoubleComplex *PsiProbeInit_d,
                                   const double * qyaReduce_d,
                                   const double * qxaReduce_d,
                                   const size_t *yBeams_d,
                                   const size_t *xBeams_d,
                                   const double yp,
                                   const double xp,
                                   const double yTiltShift,
                                   const double xTiltShift,
                                   const size_t dimi,
                                   const size_t numBeams){
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < numBeams) {
		size_t yB = yBeams_d[idx];
		size_t xB = xBeams_d[idx];
		hipDoubleComplex xp_cx = make_hipDoubleComplex(xp, 0);
		hipDoubleComplex yp_cx = make_hipDoubleComplex(yp, 0);
		hipDoubleComplex xTiltShift_cx = make_hipDoubleComplex(xTiltShift, 0);
		hipDoubleComplex yTiltShift_cx = make_hipDoubleComplex(yTiltShift, 0);
		hipDoubleComplex qya = make_hipDoubleComplex(qyaReduce_d[yB * dimi + xB], 0);
		hipDoubleComplex qxa = make_hipDoubleComplex(qxaReduce_d[yB * dimi + xB], 0);
		hipDoubleComplex arg1 = hipCmul(qxa, hipCadd(xp_cx, xTiltShift_cx));
		hipDoubleComplex arg2 = hipCmul(qya, hipCadd(yp_cx, yTiltShift_cx));
		hipDoubleComplex arg = hipCadd(arg1, arg2);
		hipDoubleComplex phase_shift = exp_cx(hipCmul(minus_2pii, arg));
		phaseCoeffs[idx] = hipCmul(phase_shift, PsiProbeInit_d[yB * dimi + xB]);
	}
}

// integrate computed intensities radially
__global__ void integrateDetector(const float* psiIntensity_ds,
                                  const float* alphaInd_d,
                                  float* integratedOutput,
                                  const size_t N,
                                  const size_t num_integration_bins) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N) {
		size_t alpha = (size_t)alphaInd_d[idx];
		if (alpha <= num_integration_bins)
			atomicAdd(&integratedOutput[alpha-1], psiIntensity_ds[idx]);
	}
}

__global__ void integrateDetector(const double* psiIntensity_ds,
                                  const double* alphaInd_d,
                                  double* integratedOutput,
                                  const size_t N,
                                  const size_t num_integration_bins) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N) {
		size_t alpha = (size_t)alphaInd_d[idx];
		if (alpha <= num_integration_bins)
			atomicAdd_double(&integratedOutput[alpha-1], psiIntensity_ds[idx]);
	}
}

__global__ void integrateDetector_real(const hipFloatComplex* psi_ds,
                                  const float* alphaInd_d,
                                  hipFloatComplex* integratedOutput,
                                  const size_t N,
                                  const size_t num_integration_bins) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N) {
		size_t alpha = (size_t)alphaInd_d[idx];
		if (alpha <= num_integration_bins)
			atomicAdd(&integratedOutput[alpha-1].x, hipCrealf(psi_ds[idx]));
	}
}

__global__ void integrateDetector_imag(const hipFloatComplex* psi_ds,
                                  const float* alphaInd_d,
                                  hipFloatComplex* integratedOutput,
                                  const size_t N,
                                  const size_t num_integration_bins) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N) {
		size_t alpha = (size_t)alphaInd_d[idx];
		if (alpha <= num_integration_bins)
			atomicAdd(&integratedOutput[alpha-1].y, hipCimagf(psi_ds[idx]));
	}
}

__global__ void integrateDetector_real(const hipDoubleComplex* psi_ds,
                                  const double* alphaInd_d,
                                  hipDoubleComplex* integratedOutput,
                                  const size_t N,
                                  const size_t num_integration_bins) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N) {
		size_t alpha = (size_t)alphaInd_d[idx];
		if (alpha <= num_integration_bins)
			atomicAdd_double(&integratedOutput[alpha-1].x, hipCreal(psi_ds[idx]));
	}
}

__global__ void integrateDetector_imag(const hipDoubleComplex* psi_ds,
                                  const double* alphaInd_d,
                                  hipDoubleComplex* integratedOutput,
                                  const size_t N,
                                  const size_t num_integration_bins) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N) {
		size_t alpha = (size_t)alphaInd_d[idx];
		if (alpha <= num_integration_bins)
			atomicAdd_double(&integratedOutput[alpha-1].y, hipCimag(psi_ds[idx]));
	}
}

__global__ void DPC_numerator_reduce(const float* psiIntensity_ds,
									 const float* q_coord,
									 float* numerator,
									 const size_t N){
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N){
		atomicAdd(&numerator[0], psiIntensity_ds[idx]*q_coord[idx]);
	}
}

__global__ void DPC_numerator_reduce(const double* psiIntensity_ds,
 									 const double* q_coord,
									 double* numerator,
									 const size_t N){
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
		if (idx < N){
			atomicAdd_double(&numerator[0], psiIntensity_ds[idx]*q_coord[idx]);
		}
}

__global__ void DPC_denominator_reduce(const float* psiIntensity_ds,
									   float* denominator,
									   const size_t N){
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N){
		atomicAdd(&denominator[0], psiIntensity_ds[idx]);
	}										   
}

__global__ void DPC_denominator_reduce(const double* psiIntensity_ds,
									   double* denominator,
									   const size_t N){
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N){
		atomicAdd_double(&denominator[0], psiIntensity_ds[idx]);
	}			
}

size_t getNextPower2(const size_t& val){
	size_t p = 0;
	while (pow(2,p) <= val)++p;
	return p;
}